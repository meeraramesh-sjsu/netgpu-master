#include "hip/hip_runtime.h"
// RabinKarpStringMatching.cpp : Defines the entry point for the console application.
//

#include<stdio.h>
#include<iostream>
using namespace std;

#define cudaAssert(f) \
		do {	\
			hipError_t err=f;\
			if(err != hipSuccess) { \
				fprintf(stderr,"hipError_t at %s:%d: %s\n",__FILE__,__LINE__,hipGetErrorString(err));\
				exit(-1);\
			}\
		}while(0)

__device__ int memCmpDev(char *input, char *pattern, int offset,int N,int M)
{
		bool result = true;
		int j = 0;
		for (int i = offset; i < offset + M && result; i++)
		{
			if (input[i] != pattern[j++]) result = false;
		}
		return !result;
}

__global__ void findIfExistsCu(char* input, int  N, char* pattern, int M,int patHash,int* result)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if(x<=N-M)
	{
		int hy,i;
		for(int hy=i=0;i<M;i++)
			hy = (hy * 256 + input[i+x]) % 997;
		if(hy == patHash && memCmpDev(input,pattern,x,N,M) == 0)
			result[x]=1;
	}
}

int main()
{
	char input[] = "HEABAL";
	char pattern[] = "AB";
	int M = 2;
	int patHash = 0;
	int N = 6;
	char* d_input;
	char* d_pattern;
	int* d_result;
	int* result;
	for (int i = 0; i < M; i++)
	{
		patHash = (patHash * 256 + pattern[i]) % 997;
	}		
	result = (int *) malloc((N-M)*sizeof(int));
	cudaAssert(hipMalloc((void **)&d_input, N * sizeof(char)));
	cudaAssert(hipMalloc((void **)&d_pattern, M * sizeof(char)));
	cudaAssert(hipMalloc((void **)&d_result,(N-M)*sizeof(int)));
	cudaAssert(hipMemcpy(d_input, input, N * sizeof(char), hipMemcpyHostToDevice));
	cudaAssert(hipMemcpy(d_pattern, pattern, M * sizeof(char), hipMemcpyHostToDevice));
	cudaAssert(hipMemset(d_result,0,(N - M)*sizeof(int)));
	dim3 block(N, 0, 0);
	dim3 grid(1, 0, 0);
	findIfExistsCu <<<grid, block>>> (d_input,N,d_pattern,M,patHash,d_result);
	cudaAssert(hipDeviceSynchronize());
	hipMemcpy(result, d_result, (N-M)*sizeof(int), hipMemcpyDeviceToHost);
	
	for(int i=0;i<=N-M;i++)
		cout << result[i]<<" ";
	return 0;
}
