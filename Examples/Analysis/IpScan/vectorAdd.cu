#include "hip/hip_runtime.h"
#include<stdio.h>

__global__ void vecAdd(int *a,int *b,int *c)
{
	int i = threadIdx.x;
	c[i]=a[i]+b[i];
}

int main()
{
	int a[10] = {0,1,2,3,4,5,6,7,8,9};
	int b[10] = {0,1,2,3,4,5,6,7,8,9};
	int *d_a, *d_b, *d_c;
	hipMalloc((void**)&d_a,10 * sizeof(int));
	hipMalloc((void**)&d_b, 10 * sizeof(int));
	hipMalloc((void**)&d_c, 10 * sizeof(int));
	hipMemcpy(d_a,a,10 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b,b, 10 * sizeof(int), hipMemcpyHostToDevice);
	vecAdd<<<1,10>>>(a,b,c);
	hipMemcpy(c,d_c,10 * sizeof(int), hipMemcpyDeviceToHost);
	for(int i=0;i<10;i++)
		printf("%d",c[i]);
}

