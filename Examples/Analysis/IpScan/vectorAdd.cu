#include "hip/hip_runtime.h"
#include<stdio.h>

__global__ vecAdd(int *a,int *b,int *c)
{
	int i = threadIdx.x;
	c[i]=a[i]+b[i];
}

int main()
{
	int a[10] = {0,1,2,3,4,5,6,7,8,9};
	int b[10] = {0,1,2,3,4,5,6,7,8,9};
	int *d_a, *d_b, *d_c;
	hipMalloc((void**)&d_a,10 * sizeof(int));
	hipMalloc((void**)&d_b, 10 * sizeof(int));
	hipMalloc((void**)&d_c, 10 * sizeof(int));
	hipMemcpy(d_a,a,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,hipMemcpyHostToDevice);
	vecAdd<<<1,10>>>(a,b,c);
	hipMemcpy(c,d_c,hipMemcpyDeviceToHost);
	for(int i=0;i<10;i++)
		printf("%d",c[i]);
}

